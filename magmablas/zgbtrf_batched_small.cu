#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @author Ahmad Abdelfattah
       @author Stan Tomov

       @precisions normal z -> s d c
*/

#include "magma_internal.h"
#include "magma_templates.h"
#include "batched_kernel_param.h"

// use this so magmasubs will replace with relevant precision, so we can comment out
// the switch case that causes compilation failure
#define PRECISION_z

#ifdef MAGMA_HAVE_HIP
#define NTCOL(M)        (max(1,64/(M)))
#else
#define NTCOL(M)        (max(1,64/(M)))
#endif

#define SLDAB(MBAND)    ((MBAND)+1)
#define sAB(i,j)        sAB[(j)*sldab + (i)]
#define dAB(i,j)        dAB[(j)*lddab + (i)]

void
zgbtrf_batched_kernel_small_sm(
    magma_int_t m, magma_int_t n,
    magma_int_t kl, magma_int_t kl,
    magmaDoubleComplex** dAB_array, int lddab,
    magma_int_t** ipiv_array, magma_int_t *info_array,
    int batchCount)
{
    extern __shared__ magmaDoubleComplex zdata[];
    const int tx  = threadIdx.x;
    const int ty  = threadIdx.y;
    const int ntx = blockDim.x;
    const int batchid = blockIdx.x * blockDim.y + ty;
    if(batchid >= batchCount) return;

    const int minmn   = min(m,n);
    const int kv      = kl + ku;
    const int mband   = (kl + 1 + kv);
    const int sldab   = SLDAB(mband);
    const int sldab_1 = sldab-1;

    magmaDoubleComplex* dAB = dAB_array[batchid];
    int linfo = 0;

    // shared memory pointers
    magmaDoubleComplex *sAB = (magmaDoubleComplex*)(zdata);
    double* dsx             = (double*)(sAB + blockDim.y * n * sldab);
    int* sipiv              = (int*)(dsx + blockDim.y * (kl+1));
    sAB   += ty * n * sldab;
    dsx   += ty * (kl+1);
    sipiv += ty * minmn;

    // init sAB
    for(int i = tx; i < n*sldab; i+=ntx) {
        sAB[i] = MAGMA_Z_ZERO;
    }
    __syncthreads();

    // read
    for(int j = 0; j < n; j++) {
        int col_start = kl + max(ku-j,0);
        int col_end   = kl + ku + min(kl, n-1-j);
        for(int i = tx + col_start; i <= col_end; i+=ntx) {
            sAB(i,j) = dAB(i,j);
        }
    }
    __syncthreads();

    int ju = 0;
    for(int j = 0; j < minmn; j++) {
        // izamax
        int km = 1 + min( kl, m-j ); // diagonal and subdiagonal(s)
        if(tx < km) {
            dsx[ tx ] = fabs(MAGMA_Z_REAL( sAB(kv+tx,j) )) + fabs(MAGMA_Z_IMAG( sAB(kv+tx,j) ));
        }
        __syncthreads();

        double rx_abs_max = dsx[0];
        int    jp       = 0;
        for(int i = 1; i < km; i++) {
            if( dsx[i] > rx_abs_max ) {
                rx_abs_max = dsx[i];
                jp         = i;
            }
        }

        linfo  = ( rx_abs_max == MAGMA_D_ZERO && linfo == 0) ? (j+1) : linfo;

        if(tx == 0) {
            sipiv[j] = jp + j + 1;    // +1 for fortran indexing
        }

        ju = max(ju, min(j+ku+jp, n-1));
        int swap_len = ju - j + 1;

        // swap
        if( !(jp == 0) ) {
            magmaDouobleComplex tmp;
            magmaDoubleComplex *sR1 = &sAB(kv   ,j);
            magmaDoubleComplex *sR2 = &sAB(kv+jp,j);
            for(int i = tx; i < swap_len; i+=ntx) {
                tmp              = sR1[i * sldab_1];
                sR1[i * sldab_1] = sR2[i * sldab_1];
                sR1[i * sldab_1] = tmp;
            }
        }
        __syncthreads();

        // scal
        magmaDoubleComplex reg = ( rx_abs_max == MAGMA_D_ZERO ) ? MAGMA_ZONE : MAGMA_Z_DIV(MAGMA_Z_ONE, sAB(kv,j) );
        for(int i = tx; i < (km-1); i+=ntx) {
            sAB(kv+1+i, j) *= reg;
        }
        __syncthreads();

        // ger
        reg = ( rx_abs_max == MAGMA_D_ZERO ) ? MAGMA_Z_ZERO : MAGMA_Z_ONE;
        magmaDoubleComplex *sV = &sAB(kv,j);
        if( tx > 0 && tx < (km-1) ) {
            for(int jj = 1; jj < swap_len; jj++) {
                sV[jj * (sldab-1) + tx] -= sV[tx] * sAB[jj * (sldab-1) + 0] * reg;
            }
        }
        __syncthreads();
    }

    // write info
    if(tx == 0) info_array[batchid] = linfo;

    // write pivot
    magma_int_t* ipiv = ipiv_array[batchid];
    for(int i = tx; i < minmn; i+=ntx) {
        ipiv[i] = sipiv[i];
    }

    // write AB
    for(int j = 0; j < n; j++) {
        for(int i = tx; i <= mband; i+=ntx) {
            dAB(i,j) = sAB(i,j);
        }
    }
}

/***************************************************************************//**
    Purpose
    -------
    zgbtrf_batched computes the LU factorization of a square N-by-N matrix A
    using partial pivoting with row interchanges.
    This routine can deal only with square matrices of size up to 32

    The factorization has the form
        A = P * L * U
    where P is a permutation matrix, L is lower triangular with unit
    diagonal elements (lower trapezoidal if m > n), and U is upper
    triangular (upper trapezoidal if m < n).

    This is the right-looking Level 3 BLAS version of the algorithm.

    This is a batched version that factors batchCount M-by-N matrices in parallel.
    dAB, ipiv, and info become arrays with one entry per matrix.

    Arguments
    ---------
    @param[in]
    n       INTEGER
            The size of each matrix A.  N >= 0.

    @param[in,out]
    dAB_array    Array of pointers, dimension (batchCount).
            Each is a COMPLEX_16 array on the GPU, dimension (LDDAB,N).
            On entry, each pointer is an M-by-N matrix to be factored.
            On exit, the factors L and U from the factorization
            A = P*L*U; the unit diagonal elements of L are not stored.

    @param[in]
    lddab    INTEGER
            The leading dimension of each array A.  LDDAB >= max(1,M).

    @param[out]
    ipiv_array  Array of pointers, dimension (batchCount), for corresponding matrices.
            Each is an INTEGER array, dimension (min(M,N))
            The pivot indices; for 1 <= i <= min(M,N), row i of the
            matrix was interchanged with row IPIV(i).

    @param[out]
    info_array  Array of INTEGERs, dimension (batchCount), for corresponding matrices.
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value
                  or another error occured, such as memory allocation failed.
      -     > 0:  if INFO = i, U(i,i) is exactly zero. The factorization
                  has been completed, but the factor U is exactly
                  singular, and division by zero will occur if it is used
                  to solve a system of equations.

    @param[in]
    batchCount  INTEGER
                The number of matrices to operate on.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_getrf_batched
*******************************************************************************/
extern "C" magma_int_t
magma_zgbtrf_batched_small(
    magma_int_t m,  magma_int_t n,
    magma_int_t kl, magma_int_t ku,
    magmaDoubleComplex** dAB_array, magma_int_t lddab,
    magma_int_t** ipiv_array, magma_int_t* info_array,
    magma_int_t batchCount, magma_queue_t queue )
{
    magma_int_t arginfo = 0;
    magma_int_t kv      = kl + ku;
    magma_int_t mband   = kv + 1 + kl;

    if( m < 0 )
        arginfo = -1;
    else if ( n < 0 )
        arginfo = -2;
    else if ( kl < 0 )
        arginfo = -3;
    else if ( ku < 0 )
        arginfo = -4;
    else if ( lddab < (kl+kv+1) )
        arginfo = -6;

    if (arginfo != 0) {
        magma_xerbla( __func__, -(arginfo) );
        return arginfo;
    }

    if( m == 0 || n == 0 ) return 0;

    magma_int_t nthreads = kl + 1;
    magma_int_t sldab    = SLDAB(mband);
    magma_int_t ntcol    = 1;   //NTCOL(nthreads);

    magma_int_t shmem  = 0;
    shmem += sldab * n * sizeof(magmaDoubleComplex); // sAB
    shmem += (kl + 1)  * sizeof(double);        // dsx
    shmem += min(m,n)  * sizeof(magma_int_t);   // pivot
    shmem *= ntcol;

    magma_int_t gridx = magma_ceildiv(batchCount, ntcol);
    dim3 threads(nthreads, ntcol, 1);
    dim3 grid(gridx, 1, 1);

    zgbtrf_batched_kernel_small_sm<<<grid, threads, shmem, queue->hip_stream()>>>
    (m, n, kl, ku, dAB_array, lddab, ipiv_array, info_array, batchCount);

    return arginfo;
}
