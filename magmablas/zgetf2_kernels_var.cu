/*
    -- MAGMA (version 2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @author Azzam Haidar
       @author Tingxing Dong
       @author Ahmad Abdelfattah

       @precisions normal z -> s d c
*/

#include "magma_internal.h"
#include "batched_kernel_param.h"
#include "magma_templates.h"
#include "shuffle.cuh"
#include "zgetf2_devicefunc.cuh"

#define PRECISION_z
