#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @author Azzam Haidar
       @author Tingxing Dong
       @author Ahmad Abdelfattah

       @precisions normal z -> s d c
*/

#include "magma_internal.h"
#include "batched_kernel_param.h"
#include "magma_templates.h"
#include "shuffle.cuh"
#include "zgetf2_devicefunc.cuh"

#define PRECISION_z

/******************************************************************************/
__global__ void
izamax_kernel_vbatched(
        int length, magma_int_t *M, magma_int_t *N,
        magmaDoubleComplex **dA_array, int Ai, int Aj, magma_int_t *ldda,
        magma_int_t** ipiv_array, int ipiv_i,
        magma_int_t *info_array, int step, int gbstep )
{
    extern __shared__ double sdata[];

    const int batchid = blockIdx.x;

    // compute the actual length
    int my_M    = (int)M[batchid];
    int my_N    = (int)N[batchid];
    int my_ldda = (int)ldda[batchid];
    // check if offsets produce out-of-bound pointers
    if( my_M <= Ai || my_N <= Aj ) return;

    // compute the length of the vector for each matrix
    my_M -= Ai;
    my_M  = min(my_M, length);

    magmaDoubleComplex *dA = dA_array[batchid] + Aj * my_ldda + Ai;
    magma_int_t *ipiv = ipiv_array[batchid] + ipiv_i;
    int tx = threadIdx.x;

    double *shared_x = sdata;
    int *shared_idx = (int*)(shared_x + zamax);

    izamax_devfunc(my_M, dA, 1, shared_x, shared_idx);

    if (tx == 0) {
        *ipiv = shared_idx[0] + step + 1; // Fortran Indexing & adjust pivot
        if (shared_x[0] == MAGMA_D_ZERO) {
            info_array[batchid] = shared_idx[0] + step + gbstep + 1;
        }
    }
}

/******************************************************************************/
extern "C" magma_int_t
magma_izamax_vbatched(
        magma_int_t length, magma_int_t *M, magma_int_t *N,
        magmaDoubleComplex **dA_array, magma_int_t Ai, magma_int_t Aj, magma_int_t* ldda,
        magma_int_t** ipiv_array, magma_int_t ipiv_i,
        magma_int_t *info_array, magma_int_t step, magma_int_t gbstep,
        magma_int_t batchCount, magma_queue_t queue)
{
    dim3 grid(batchCount, 1, 1);
    dim3 threads(zamax, 1, 1);

    izamax_kernel_vbatched<<< grid, threads, zamax * (sizeof(double) + sizeof(int)), queue->hip_stream() >>>
    (length, M, N, dA_array, Ai, Aj, ldda, ipiv_array, ipiv_i, info_array, step, gbstep );

    return 0;
}

/******************************************************************************/
__global__
void zswap_kernel_vbatched(
        int max_n, magma_int_t *M, magma_int_t *N,
        magmaDoubleComplex **dA_array, int Ai, int Aj, magma_int_t* ldda,
        magma_int_t** ipiv_array, int piv_adjustment)
{
    const int batchid = blockIdx.x;
    const int my_ldda = (int)ldda[batchid];
    int my_M          = (int)M[batchid];
    int my_N          = (int)N[batchid];
    int my_minmn      = min(my_M, my_N);

    // check if offsets produce out-of-bound pointers
    if( my_M <= Ai || my_N <= Aj || my_minmn <= Ai ) return;

    my_N -= Aj; // this is the maximum possible width
    my_N = min(my_N, max_n);

    // read the pivot entry at Ai
    magma_int_t *ipiv = ipiv_array[batchid] + Ai;
    __shared__ int jp;
    if (threadIdx.x == 0){
        jp  = ipiv[0] - 1; // roll-back Fortran indexing
        // magma_izamax_vbatched adjusts the pivot, so roll it back
        // because Ai and Aj are offsets that already take care of that
        jp -= piv_adjustment;
    }
    __syncthreads();

    if (jp == 0) return; // no swapping required

    magmaDoubleComplex *dA  = dA_array[batchid] + Aj * my_ldda + Ai;
    magmaDoubleComplex *dA1 = dA;
    magmaDoubleComplex *dA2 = dA + jp;

    zswap_device_v2(my_N, dA1, my_ldda, dA2, my_ldda );
}

/******************************************************************************/
extern "C" magma_int_t
magma_zswap_vbatched(
        magma_int_t max_n, magma_int_t *M, magma_int_t *N,
        magmaDoubleComplex **dA_array, magma_int_t Ai, magma_int_t Aj, magma_int_t *ldda,
        magma_int_t** ipiv_array, magma_int_t piv_adjustment,
        magma_int_t batchCount, magma_queue_t queue)
{
    dim3 grid(batchCount, 1, 1);
    dim3 threads(zamax, 1, 1);

    zswap_kernel_vbatched<<< grid, threads, 0, queue->hip_stream() >>>
    (max_n, M, N, dA_array, Ai, Aj, ldda, ipiv_array, piv_adjustment);

    return 0;
}

/******************************************************************************/
__global__
void zscal_zgeru_1d_generic_kernel_vbatched(
        int max_m, int max_n,
        magma_int_t *M, magma_int_t *N,
        magmaDoubleComplex **dA_array, int Ai, int Aj, magma_int_t *ldda,
        magma_int_t *info_array, int step, int gbstep)
{
    const int batchid = blockIdx.z;
    int my_M    = (int)M[batchid];
    int my_N    = (int)N[batchid];
    int my_ldda = (int)ldda[batchid];

    if( my_M <= Ai || my_N <= Aj ) return;
    my_M -= Ai; // this is the largest possible m per matrix
    my_N -= Aj; // this is the largest possible n per matrix

    my_M = min(my_M, max_m);
    my_N = min(my_N, max_n);

    magmaDoubleComplex* dA = dA_array[batchid] + Aj * my_ldda + Ai;
    magma_int_t *info = &info_array[batchid];
    zscal_zgeru_generic_device(my_M, my_N, dA, my_ldda, info, step, gbstep);
}


/******************************************************************************/
extern "C"
magma_int_t magma_zscal_zgeru_vbatched(
        magma_int_t max_M, magma_int_t max_N,
        magma_int_t *M, magma_int_t *N,
        magmaDoubleComplex **dA_array, magma_int_t Ai, magma_int_t Aj, magma_int_t *ldda,
        magma_int_t *info_array, magma_int_t step, magma_int_t gbstep,
        magma_int_t batchCount, magma_queue_t queue)
{
    /*
    Specialized kernel which merged zscal and zgeru the two kernels
    1) zscale the first column vector A(1:M-1,0) with 1/A(0,0);
    2) Performe a zgeru Operation for trailing matrix of A(1:M-1,1:N-1) += alpha*x*y**T, where
       alpha := -1.0; x := A(1:M-1,0) and y:= A(0,1:N-1);
    */

    magma_int_t max_batchCount = queue->get_maxBatch();
    const int tbx = 256;
    dim3 threads(tbx, 1, 1);

    for(magma_int_t i = 0; i < batchCount; i+=max_batchCount) {
        magma_int_t ibatch = min(max_batchCount, batchCount-i);
        dim3 grid(magma_ceildiv(max_M,tbx), 1, ibatch);

        zscal_zgeru_1d_generic_kernel_vbatched<<<grid, threads, 0, queue->hip_stream()>>>
        (max_M, max_N, M+i, N+i, dA_array+i, Ai, Aj, ldda+i, info_array+i, step, gbstep);
    }
    return 0;
}

/******************************************************************************/
#define dA(i,j)              sA[(j) * my_ldda + (i)]
#define sA(i,j)              sA[(j) * my_M + (i)]
__global__
void
zgetf2_fused_sm_kernel_vbatched(
        int max_M, int max_N, int max_minMN, int max_MxN,
        magma_int_t *M, magma_int_t *N,
        magmaDoubleComplex** dA_array, int Ai, int Aj, magma_int_t* ldda,
        magma_int_t** dipiv_array, int ipiv_i,
        magma_int_t *info,  int gbstep, int batchCount )
{
    extern __shared__ magmaDoubleComplex sdata[];
    const int tx      = threadIdx.x;
    const int ty      = threadIdx.y;
    const int ntx     = blockDim.x;
    const int batchid = (blockIdx.x * blockDim.y) + ty;
    if(batchid >= batchCount) return;

    // read data of assigned problem
    const int my_m         = (int)M[batchid];
    const int my_N         = (int)N[batchid];
    const int my_ldda      = (int)ldda[batchid];
    const int my_minmn     = min(my_M, my_N);
    magmaDoubleComplex* dA = dA_array[batchid] + Aj * my_ldda + Ai;
    magma_int_t* dipiv     = dipiv_array[batchid] + ipiv_i;

    // check offsets
    if( my_M <= Ai || my_N <= Aj || my_minmn <= ipiv_i ) return;
    my_M     -= Ai;
    my_N     -= Aj;
    my_minmn  = min(my_M, my_N);

    magmaDoubleComplex *sA = (magmaDoubleComplex*)(sdata);
    double* dsx = (double*)(sA + blockDim.y * max_MxN);
    int* isx    = (int*)(dsx + blockDim.y * max_M);
    int* sipiv  = (int*)(isx + blockDim.y * max_M);
    dsx   += ty * max_M;
    isx   += ty * max_M;
    sipiv += ty * max_minMN;

    magmaDoubleComplex reg  = MAGMA_Z_ZERO;
    magmaDoubleComplex rTmp = MAGMA_Z_ZERO;

    int max_id, rowid = tx;
    int linfo = (gbstep == 0) ? 0 : *info;
    double rx_abs_max = MAGMA_D_ZERO;

    // init sipiv
    for(int i = tx; i < my_minmn; i+=ntx) {
        sipiv[i] = 0;
    }

    // read
    for(int j = 0; j < my_N; j++){
        for(int i = tx; i < my_M; i+=ntx) {
            sA(i,j) = dA(i,j);
        }
    }
    __syncthreads();

    for(int j = 0; j < my_minmn; j++){
        // izamax and find pivot
        for(int i = tx; i < my_M-j; i+=ntx) {
            dsx[ i ] = fabs(MAGMA_Z_REAL( sA(i,j) )) + fabs(MAGMA_Z_IMAG( sA(i,j) ));
            isx[ i ] = i;
        }
        __syncthreads();
        magma_getidmax_n(my_M-j, tx, dsx, isx);
        // the above devfunc has syncthreads at the end
        rx_abs_max = dsx[0];
        max_id     = isx[0];
        linfo  = ( rx_abs_max == MAGMA_D_ZERO && linfo == 0) ? (gbstep+j+1) : linfo;
        if( tx == 0 ) sipiv[ j ] = max_id;
        __syncthreads();

        // swap
        if(max_id != j) {
            for(int i = tx; i < my_N; i+=ntx) {
                reg          = sA(j     ,i);
                sA(i,j)      = sA(max_id,i);
                sA(max_id,i) = reg;
            }
        }
        __syncthreads();

        if( linfo == 0 ) {
            reg = MAGMA_Z_DIV( MAGMA_Z_ONE, sA(j,j) );
            for(int i = (tx+j+1); i < my_M; i+=ntx) {
                rTmp    = reg * sA(i,j);
                sA(i,j) = rTmp;
                for(int jj = j+1; jj < my_N; jj++) {
                    sA(i,jj) -= rTmp * sA(j,jj);
                }
            }
        }
        __syncthreads();
    }

    if(tx == 0){
        (*info) = (magma_int_t)( linfo );
    }

    // write pivot
    for(int i = tx; i < my_minmn; i+=ntx) {
        dipiv[i] = (magma_int_t)(sipiv[i]);
    }

    // write A
    for(int j = 0; j < my_N; j++) {
        for(int i = tx; i < my_M; i+=ntx) {
            dA(i,j) = sA(i,j);
        }
    }
}

////////////////////////////////////////////////////////////////////////////////
extern "C" magma_int_t
magma_zgetf2_fused_sm_vbatched(
    magma_int_t max_M, magma_int_t max_N, magma_int_t max_minMN, magma_int_t max_MxN,
    magma_int_t* m, magma_int_t* n,
    magmaDoubleComplex** dA_array, magma_int_t Ai, magma_int_t Aj, magma_int_t* ldda,
    magma_int_t** dipiv_array, magma_int_t ipiv_i,
    magma_int_t* info_array,
    magma_int_t nthreads, magma_int_t check_launch_only,
    magma_int_t batchCount, magma_queue_t queue )
{
    magma_int_t arginfo = 0;
    magma_device_t device;
    magma_getdevice( &device );

    nthreads = nthreads <= 0 ? (max_M/2) : nthreads;
    #ifdef MAGMA_HAVE_CUDA
    nthreads = magma_roundup(nthreads, 32);
    #else
    nthreads = magma_roundup(nthreads, 64);
    #endif
    nthreads = min(nthreads, 1024);

    // in a variable-size setting, setting ntcol > 1 may lead to
    // kernel deadlocks due to different thread-groups calling
    // syncthreads at different points
    const magma_int_t ntcol = 1;
    magma_int_t shmem = ( max_MxN   * sizeof(magmaDoubleComplex) );
    shmem            += ( max_M     * sizeof(double) );
    shmem            += ( max_M     * sizeof(int) );
    shmem            += ( max_minMN * sizeof(int) );
    shmem            *= ntcol;
    magma_int_t gridx = magma_ceildiv(batchCount, ntcol);
    dim3 grid(gridx, 1, 1);
    dim3 threads( nthreads, ntcol, 1);

    // get max. dynamic shared memory on the GPU
    magma_int_t nthreads_max, shmem_max = 0;
    hipDeviceGetAttribute (&nthreads_max, hipDeviceAttributeMaxThreadsPerBlock, device);
    #if CUDA_VERSION >= 9000
    hipDeviceGetAttribute (&shmem_max, hipDeviceAttributeSharedMemPerBlockOptin, device);
    if (shmem <= shmem_max) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(zgetf2_fused_sm_kernel_vbatched), hipFuncAttributeMaxDynamicSharedMemorySize, shmem);
    }
    #else
    hipDeviceGetAttribute (&shmem_max, hipDeviceAttributeMaxSharedMemoryPerBlock, device);
    #endif    // CUDA_VERSION >= 9000

    magma_int_t total_threads = nthreads * ntcol;
    if ( total_threads > nthreads_max || shmem > shmem_max ) {
        // printf("error: kernel %s requires too many threads or too much shared memory\n", __func__);
        arginfo = -100;
        return arginfo;
    }

    if( check_launch_only == 1 ) return arginfo;

    void *kernel_args[] = {&max_M, &max_N, &max_minMN, &max_MxN, &m, &n, &dA_array, &Ai, &Aj, &ldda, &dipiv_array, &ipiv_i, &info, &gbstep, &batchCount};
    hipError_t e = hipLaunchKernel((void*)zgetf2_fused_sm_kernel_vbatched, grid, threads, kernel_args, shmem, queue->hip_stream());
    if( e != hipSuccess ) {
        // printf("error in %s : failed to launch kernel %s\n", __func__, hipGetErrorString(e));
        arginfo = -100;
    }

    return arginfo;
}


