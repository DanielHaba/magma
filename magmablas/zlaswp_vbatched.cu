#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c

       @author Ahmad Abdelfattah
*/
#include "magma_internal.h"
#include "batched_kernel_param.h"

#define BLK_SIZE 256
#define ZLASWP_COL_NTH 32


/******************************************************************************/
// serial swap that does swapping one row by one row
// this is the vbatched routine, for swapping to the left of the panel
__global__ void zlaswp_left_rowserial_kernel_vbatched(
                int n, int nb,
                magma_int_t *M, magma_int_t *N,
                magmaDoubleComplex **dA_array, int Ai, int Aj, magma_int_t *ldda,
                int k1, int k2,
                magma_int_t** ipiv_array )
{
    const int batchid = blockIdx.z;
    const int tid     = threadIdx.x + blockDim.x*blockIdx.x;

    int my_M     = (int)M[batchid];
    int my_N     = (int)N[batchid];
    int my_minmn = min(my_M, my_N);
    int my_ldda  = (int)ldda[batchid];
    magmaDoubleComplex* dA = dA_array[batchid] + Aj * my_ldda + Ai;
    magma_int_t *dipiv = ipiv_array[batchid];

    // check if offsets produce out-of-bound pointers
    if( my_M <= Ai || my_N <= Aj ) return;

    k1--;
    k2--;

    // check the offsets k1, k2
    if( k1 >= my_minmn ) return;
    k2 = min(k2, my_minmn-1);

    // check the input scalar 'n'
    const int my_max_n = (magma_ceildiv(my_minmn, nb) - 1) * nb;
    const int my_n     = min(n, my_max_n);

    if (tid < my_n) {
        magmaDoubleComplex A1;

        for (int i1 = k1; i1 < k2; i1++) {
            int i2 = dipiv[i1] - 1;  // Fortran index, switch i1 and i2
            if ( i2 != i1 ) {
                A1 = dA[i1 + tid * my_ldda];
                dA[i1 + tid * my_ldda] = dA[i2 + tid * my_ldda];
                dA[i2 + tid * my_ldda] = A1;
            }
        }
    }
}

/******************************************************************************/
// serial swap that does swapping one row by one row
// this is the vbatched routine, for swapping to the right of the panel
__global__ void zlaswp_right_rowserial_kernel_vbatched(
                int n,
                magma_int_t *M, magma_int_t *N,
                magmaDoubleComplex **dA_array, int Ai, int Aj, magma_int_t *ldda,
                int k1, int k2,
                magma_int_t** ipiv_array )
{
    const int batchid = blockIdx.z;
    const int tid     = threadIdx.x + blockDim.x*blockIdx.x;

    int my_M     = (int)M[batchid];
    int my_N     = (int)N[batchid];
    int my_minmn = min(my_M, my_N);
    int my_ldda  = (int)ldda[batchid];
    magmaDoubleComplex* dA = dA_array[batchid] + Aj * my_ldda + Ai;
    magma_int_t *dipiv = ipiv_array[batchid];

    // check if offsets produce out-of-bound pointers
    if( my_M <= Ai || my_N <= Aj ) return;

    k1--;
    k2--;

    // check the offsets k1, k2
    if( k1 >= my_minmn ) return;
    k2 = min(k2, my_minmn-1);

    // check the input scalar 'n'
    const int my_max_n = my_N - Aj;
    const int my_n     = min(n, my_max_n);

    if (tid < my_n) {
        magmaDoubleComplex A1;

        for (int i1 = k1; i1 < k2; i1++) {
            int i2 = dipiv[i1] - 1;  // Fortran index, switch i1 and i2
            if ( i2 != i1 ) {
                A1 = dA[i1 + tid * my_ldda];
                dA[i1 + tid * my_ldda] = dA[i2 + tid * my_ldda];
                dA[i2 + tid * my_ldda] = A1;
            }
        }
    }
}

/******************************************************************************/
// serial swap that does swapping one row by one row, similar to LAPACK
// K1, K2 are in Fortran indexing
extern "C" void
magma_zlaswp_left_rowserial_vbatched(
        magma_int_t n, magma_int_t nb,
        magma_int_t *M, magma_int_t *N, magmaDoubleComplex** dA_array, magma_int_t Ai, magma_int_t Aj, magma_int_t *ldda,
        magma_int_t k1, magma_int_t k2,
        magma_int_t **ipiv_array,
        magma_int_t batchCount, magma_queue_t queue)
{
    if (n == 0) return;

    magma_int_t max_batchCount  = queue->get_maxBatch();
    magma_int_t blocks          = magma_ceildiv( n, BLK_SIZE );
    magma_int_t max_BLK_SIZE__n = max(BLK_SIZE, n);

    for(magma_int_t i = 0; i < batchCount; i+=max_batchCount) {
        magma_int_t ibatch = min(max_batchCount, batchCount-i);
        dim3  grid(blocks, 1, ibatch);

        zlaswp_left_rowserial_kernel_vbatched
        <<< grid, max_BLK_SIZE__n, 0, queue->hip_stream() >>>
        (n, nb, M, N, dA_array, Ai, Aj, ldda,k1, k2, ipiv_array);
    }
}


/******************************************************************************/
// serial swap that does swapping one row by one row, similar to LAPACK
// K1, K2 are in Fortran indexing
extern "C" void
magma_zlaswp_right_rowserial_vbatched(
        magma_int_t n,
        magma_int_t *M, magma_int_t *N, magmaDoubleComplex** dA_array, magma_int_t Ai, magma_int_t Aj, magma_int_t *ldda,
        magma_int_t k1, magma_int_t k2,
        magma_int_t **ipiv_array,
        magma_int_t batchCount, magma_queue_t queue)
{
    if (n == 0) return;

    magma_int_t max_batchCount  = queue->get_maxBatch();
    magma_int_t blocks          = magma_ceildiv( n, BLK_SIZE );
    magma_int_t max_BLK_SIZE__n = max(BLK_SIZE, n);

    for(magma_int_t i = 0; i < batchCount; i+=max_batchCount) {
        magma_int_t ibatch = min(max_batchCount, batchCount-i);
        dim3  grid(blocks, 1, ibatch);

        zlaswp_right_rowserial_kernel_vbatched
        <<< grid, max_BLK_SIZE__n, 0, queue->hip_stream() >>>
        (n, M, N, dA_array, Ai, Aj, ldda,k1, k2, ipiv_array);
    }
}
